#include "hip/hip_runtime.h"
//
// Created by creator on 18-8-30.
//

#include "cide.h"
#include "common.h"

#ifndef M_PI
# define M_PI        3.14159265358979323846f    /* pi */
#endif
/**
 * turn bgr color c to xyc color
 * @param bgr a bgr color
 * @return a xyc color
 */
__device__ float3 toXYC(uchar3 bgr) {

    float xyz[3];
    float r = (bgr.z / 255.0f);//channel R
    float g = (bgr.y / 255.0f);//channel G
    float b = (bgr.x / 255.0f);//channel B

    if (r > 0.04045)
        r = (float) pow(((r + 0.055) / 1.055), 2.4);
    else
        r = r / 12.92f;

    if (g > 0.04045)
        g = (float) pow(((g + 0.055) / 1.055), 2.4);
    else
        g = g / 12.92f;

    if (b > 0.04045)
        b = (float) pow(((b + 0.055) / 1.055), 2.4);
    else
        b = b / 12.92f;

    r *= 100;
    g *= 100;
    b *= 100;

    //Observer. = 2°, Illuminant = D65
    xyz[0] = r * 0.4124f + g * 0.3576f + b * 0.1805f;
    xyz[1] = r * 0.2126f + g * 0.7152f + b * 0.0722f;
    xyz[2] = r * 0.0193f + g * 0.1192f + b * 0.9505f;

    float3 va;
    va.x = xyz[0];//channel X
    va.y = xyz[1];//channel Y
    va.z = xyz[2];//channel Z
    return va;
}

/**
 * turn bgr color to a lab color
 * @param bgr a bgr color
 * @return a lab color
 */
__device__ LAB toLAB(uchar3 bgr) {
    float3 xyz = toXYC(bgr);
    float lab[3];
    float x = xyz.x / 95.047f; //Observer. = 2°, Illuminant = D65
    float y = xyz.y / 100.000f;
    float z = xyz.z / 108.883f;

    if (x > 0.008856)
        x = (float) pow(x, (1.0 / 3.0));
    else
        x = (7.787f * x) + (16 / 116);

    if (y > 0.008856)
        y = (float) pow(y, (1.0 / 3.0));
    else
        y = (7.787f * y) + (16 / 116);

    if (z > 0.008856)
        z = (float) pow(z, (1.0 / 3.0));
    else
        z = (7.787f * z) + (16 / 116);

    lab[0] = (116 * y) - 16;
    lab[1] = 500 * (x - y);
    lab[2] = 200 * (y - z);

    lab[0] = lab[0] < 0.0 ? 0.0f : lab[0];

    LAB va;
    va.l = lab[0];//channel L
    va.a = lab[1];//channel A
    va.b = lab[2];//channel B
    return va;
}

__device__ float deg2Rad(const float deg) {
    return (deg * (M_PI / 180.0f));
}

__device__ float cide_distance(uchar3 bgr1, uchar3 bgr2) {
    LAB lab1 = toLAB(bgr1);
    LAB lab2 = toLAB(bgr2);

    const float k_L = 1.0, k_C = 1.0, k_H = 1.0;
    const float deg360InRad = deg2Rad(360.0);
    const float deg180InRad = deg2Rad(180.0);
    const float pow25To7 = 6103515625.0f; /* pow(25, 7) */

    /*
	 * Step 1
	 */
    /* Equation 2 */
    float C1 = (float) sqrt((lab1.a * lab1.a) + (lab1.b * lab1.b));
    float C2 = (float) sqrt((lab2.a * lab2.a) + (lab2.b * lab2.b));
    /* Equation 3 */
    float barC = (C1 + C2) / 2.0f;
    /* Equation 4 */
    float G = 0.5f * (float) (1 - sqrt(pow(barC, 7) / (pow(barC, 7) + pow25To7)));
    /* Equation 5 */
    float a1Prime = (1.0f + G) * lab1.a;
    float a2Prime = (1.0f + G) * lab2.a;
    /* Equation 6 */
    float CPrime1 = (float) sqrt((a1Prime * a1Prime) + (lab1.b * lab1.b));
    float CPrime2 = (float) sqrt((a2Prime * a2Prime) + (lab2.b * lab2.b));
    /* Equation 7 */
    float hPrime1;
    if (lab1.b == 0 && a1Prime == 0)
        hPrime1 = 0.0;
    else {
        hPrime1 = (float) atan2(lab1.b, a1Prime);
        /*
         * This must be converted to a hue angle in degrees between 0
         * and 360 by addition of 2􏰏 to negative hue angles.
         */
        if (hPrime1 < 0)
            hPrime1 += deg360InRad;
    }
    float hPrime2;
    if (lab2.b == 0 && a2Prime == 0)
        hPrime2 = 0.0;
    else {
        hPrime2 = (float) atan2(lab2.b, a2Prime);
        /*
         * This must be converted to a hue angle in degrees between 0
         * and 360 by addition of 2􏰏 to negative hue angles.
         */
        if (hPrime2 < 0)
            hPrime2 += deg360InRad;
    }

    /*
     * Step 2
     */
    /* Equation 8 */
    float deltaLPrime = lab2.l - lab1.l;
    /* Equation 9 */
    float deltaCPrime = CPrime2 - CPrime1;
    /* Equation 10 */
    float deltahPrime;
    float CPrimeProduct = CPrime1 * CPrime2;
    if (CPrimeProduct == 0)
        deltahPrime = 0;
    else {
        /* Avoid the fabs() call */
        deltahPrime = hPrime2 - hPrime1;
        if (deltahPrime < -deg180InRad)
            deltahPrime += deg360InRad;
        else if (deltahPrime > deg180InRad)
            deltahPrime -= deg360InRad;
    }
    /* Equation 11 */
    float deltaHPrime = 2.0f * (float) sqrt(CPrimeProduct) *
                        (float) sin(deltahPrime / 2.0f);

    /*
     * Step 3
     */
    /* Equation 12 */
    float barLPrime = (lab1.l + lab2.l) / 2.0f;
    /* Equation 13 */
    float barCPrime = (CPrime1 + CPrime2) / 2.0f;
    /* Equation 14 */
    float barhPrime, hPrimeSum = hPrime1 + hPrime2;
    if (CPrime1 * CPrime2 == 0) {
        barhPrime = hPrimeSum;
    } else {
        if (fabs(hPrime1 - hPrime2) <= deg180InRad)
            barhPrime = hPrimeSum / 2.0f;
        else {
            if (hPrimeSum < deg360InRad)
                barhPrime = (hPrimeSum + deg360InRad) / 2.0f;
            else
                barhPrime = (hPrimeSum - deg360InRad) / 2.0f;
        }
    }
    /* Equation 15 */
    float T = 1.0f - (float) (0.17f * cos(barhPrime - deg2Rad(30.0))) +
              (float) (0.24f * cos(2.0f * barhPrime)) +
              (float) (0.32f * cos((3.0f * barhPrime) + deg2Rad(6.0))) -
              (float) (0.20f * cos((4.0f * barhPrime) - deg2Rad(63.0)));
    /* Equation 16 */
    float deltaTheta = deg2Rad(30.0) *
                       (float) exp(-pow((barhPrime - deg2Rad(275.0)) / deg2Rad(25.0), 2.0));
    /* Equation 17 */
    float R_C = 2.0f * (float) sqrt(pow(barCPrime, 7.0f) /
                                    (pow(barCPrime, 7.0f) + pow25To7));
    /* Equation 18 */
    float S_L = 1 + (float) ((0.015 * pow(barLPrime - 50.0, 2.0)) /
                             sqrt(20 + pow(barLPrime - 50.0, 2.0)));
    /* Equation 19 */
    float S_C = 1 + (0.045f * barCPrime);
    /* Equation 20 */
    float S_H = 1 + (0.015f * barCPrime * T);
    /* Equation 21 */
    float R_T = (float) (-sin(2.0 * deltaTheta)) * R_C;

    /* Equation 22 */
    float deltaE = (float) sqrt(
            pow(deltaLPrime / (k_L * S_L), 2.0) +
            pow(deltaCPrime / (k_C * S_C), 2.0) +
            pow(deltaHPrime / (k_H * S_H), 2.0) +
            (R_T * (deltaCPrime / (k_C * S_C)) * (deltaHPrime / (k_H * S_H))));

    return deltaE;
}

/**
 * compute CIDE2000 distance between point(j, i) and it two neighbor points
 * @param src input image
 * @param out out put result
 */
__global__ void kernel_compute_distance(cv::cuda::PtrStepSz<uchar3> src,int row_index,int col_index,
                                        cv::cuda::PtrStepSz<float> out) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; //行坐标
    int j = threadIdx.y + blockIdx.y * blockDim.y; //列坐标

    if (i >= src.rows || j >= src.cols) {
        return;
    }
    uchar3 bgr1 = src(row_index, col_index);
    uchar3 bgr2 = src(i, j);
    out(i, j) = cide_distance(bgr1,bgr2);
}

void compute_distance(const cv::Mat &src,int row_index,int col_index, cv::Mat &dst) {
    cv::cuda::GpuMat g_src;
    cv::cuda::GpuMat g_dst(src.size(), CV_32FC1);
    g_src.upload(src);

    dim3 block(32, 32);
    dim3 grid((src.rows + block.y - 1) / block.y, (src.cols + block.x - 1) / block.x);
    kernel_compute_distance<<< grid, block, 0 >>>(g_src, row_index,col_index,g_dst);

    g_dst.download(dst);
}

__global__ void kernel_init(){
    printf("kernel initialized\n");
}

void init_cuda() {
    kernel_init<<<1,1,0>>>();
}